#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void multiply(int* a,int* b,int sharedDim,int bCols,int* c){
    // Row
    int row = blockIdx.x;
    // Col
    int col = threadIdx.x;
    int sm = 0;
    for (int i=0;i<sharedDim;i++){
        sm+=a[row*sharedDim+i]*b[i*bCols+col];
    }
    c[row*bCols+col] = sm;
}
int main(){
    int aRows = 1;
    int aColsBRows = 3;
    int bCols = 2;

    // Define the matrices
    int a[aRows][aColsBRows] = {{1,2,3}};
    int b[aColsBRows][bCols] = {{1,2},{1,2},{1,1}};
    int c[aRows][bCols] = {0};

    // Create memory on the GPU for them
    int* gpuA = 0;
    int* gpuB = 0;
    int* gpuC = 0;

    hipMalloc(&gpuA,sizeof(a));
    hipMalloc(&gpuB,sizeof(b));
    hipMalloc(&gpuC,sizeof(c));

    // Copy memory to GPU
    hipMemcpy(gpuA,a,sizeof(a),hipMemcpyHostToDevice);
    hipMemcpy(gpuB,b,sizeof(b),hipMemcpyHostToDevice);

    multiply<<<aRows,bCols>>>(gpuA,gpuB,aColsBRows,bCols,gpuC);

    // Get C from GPU
    hipMemcpy(c,gpuC,sizeof(c),hipMemcpyDeviceToHost);

    for (int i=0;i<aRows;i++){
        printf("|");
        for (int k=0;k<bCols;k++){
            printf("%d",c[i][k]);
            if (k<bCols-1) printf(" ");
        }
        printf("|\n");
    }
}